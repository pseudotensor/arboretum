#include "hip/hip_runtime.h"
//#include <omp.h>
#include "io.h"
#include <algorithm>
#include <functional>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <vector>

namespace arboretum {
namespace io {
using namespace std;

DataMatrix::DataMatrix(int rows, int columns)
    : rows(rows), columns(columns), columns_dense(columns), columns_sparse(0) {
  _init = false;
  data.resize(columns);
  index.resize(columns);
  index_device.resize(columns);
  data_device.resize(columns);

  for (int i = 0; i < columns; ++i) {
    data[i].resize(rows);
  }
}

void DataMatrix::Init() {
  if (!_init) {
    lil_column_device.resize(columns_sparse);

#pragma omp parallel for
    for (size_t i = 0; i < columns_dense; ++i) {
      index[i] = SortedIndex(i);
    }

    _init = true;
  }
}

std::vector<unsigned int> DataMatrix::SortedIndex(int column) {
  auto &v = data[column];
  size_t size = v.size();
  std::vector<unsigned int> idx(size);
  for (size_t i = 0; i < size; i++)
    idx[i] = i;

  sort(idx.begin(), idx.end(),
       [&v](size_t i1, size_t i2) { return v[i1] < v[i2]; });

  return idx;
}

void DataMatrix::UpdateGrad() {}
void DataMatrix::TransferToGPU(size_t free, bool verbose) {
  size_t index_size = sizeof(unsigned int) * rows;
  size_t data_size = sizeof(float) * rows;
  size_t copy_count = std::min(free / index_size, columns_dense);
  for (size_t i = 0; i < copy_count; ++i) {
    index_device[i] = index[i];
  }
  if (verbose)
    printf("copied index data %ld from %ld \n", copy_count, columns_dense);

  free -= copy_count * index_size;
  copy_count = std::min(free / data_size, columns_dense);
  for (size_t i = 0; i < copy_count; ++i) {
    data_device[i].resize(rows + 1);
    data_device[i][0] = -std::numeric_limits<float>::infinity();
    thrust::copy(data[i].begin(), data[i].end(),
                 data_device[i].begin() + 1);
  }
  if (verbose)
    printf("copied features data %ld from %ld \n", copy_count, columns_dense);

  free -= copy_count * data_size;

  copy_count = 0;

  for (size_t i = 0; i < columns_sparse; ++i) {
    size_t size = lil_column[i].size();

    if (size * sizeof(unsigned int) < free) {
      copy_count++;
      lil_column_device[i] = lil_column[i];
      free -= size * sizeof(unsigned int);
    } else {
      break;
    }
  }
  if (verbose)
    printf("copied sparse features %ld from %ld \n", copy_count,
           columns_sparse);
}
}
}
