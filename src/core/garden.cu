#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include "cuda_helpers.h"
#include "hip/hip_runtime.h"
#include "garden.h"
#include "objective.h"
#include "param.h"
#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <limits>
#include <math.h>
#include <random>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <trove/aos.h>
#include <trove/ptr.h>

namespace arboretum {
namespace core {
using namespace thrust;
using namespace thrust::cuda;
using thrust::host_vector;
using thrust::device_vector;
using thrust::cuda::experimental::pinned_allocator;

union my_atomics {
  float floats[2];              // floats[0] = maxvalue
  unsigned int ints[2];         // ints[1] = maxindex
  unsigned long long int ulong; // for atomic update
};

struct GainFunctionParameters {
  const unsigned int min_leaf_size;
  const float hess;
  const float gamma;
  const float lambda;
  const float alpha;
  GainFunctionParameters(const unsigned int min_leaf_size, const float hess,
                         const float gamma, const float lambda,
                         const float alpha)
      : min_leaf_size(min_leaf_size), hess(hess), gamma(gamma), lambda(lambda),
        alpha(alpha) {}
};

__forceinline__ __device__ unsigned long long int
updateAtomicMax(unsigned long long int *address, float val1,
                unsigned int val2) {
  my_atomics loc, loctest;
  loc.floats[0] = val1;
  loc.ints[1] = val2;
  loctest.ulong = *address;
  while (loctest.floats[0] < val1)
    loctest.ulong = atomicCAS(address, loctest.ulong, loc.ulong);
  return loctest.ulong;
}

template <class type1>
__global__ void
gather_kernel_simple(const unsigned int *const __restrict__ position,
                     const type1 *const __restrict__ in1, type1 *out1,
                     const size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    out1[i] = in1[position[i]];
  }
}

template <class T>
__global__ void gather(const unsigned int *const __restrict__ position,
                        T *in, T *out,
                       const size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    const unsigned int index = position[i];
    trove::coalesced_ptr<T> s(in);
    trove::coalesced_ptr<T> r(out);
    T data = s[index];
    r[index] = data;
  }
}

template <class type1>
__global__ void
gather_kernel_temp(const unsigned int *const __restrict__ position,
                   const type1 *const __restrict__ data, type1 *out, type1 *tmp,
                   const size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    tmp[i] = data[position[i]];
  }

  __syncthreads();

  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    out[i] = tmp[i];
  }
}

__forceinline__ __device__ __host__ float
gain_func(const double2 left_sum, const double2 total_sum,
          const size_t left_count, const size_t total_count,
          const GainFunctionParameters &params) {
  const double2 right_sum = total_sum - left_sum;
  if (left_count >= params.min_leaf_size &&
      (total_count - left_count) >= params.min_leaf_size &&
      std::abs(left_sum.y) >= params.hess &&
      std::abs(right_sum.y) >= params.hess) {
    const float l = (left_sum.x * left_sum.x) / (left_sum.y + params.lambda);
    const float r = (right_sum.x * right_sum.x) / (right_sum.y + params.lambda);
    const float p = (total_sum.x * total_sum.x) / (total_sum.y + params.lambda);
    return l + r - p;
  } else {
    return 0.0;
  }
}

__forceinline__ __device__ __host__ float
gain_func(const float2 left_sum, const float2 total_sum,
          const size_t left_count, const size_t total_count,
          const GainFunctionParameters &params) {
  const float2 right_sum = total_sum - left_sum;
  if (left_count >= params.min_leaf_size &&
      (total_count - left_count) >= params.min_leaf_size &&
      std::abs(left_sum.y) >= params.hess &&
      std::abs(right_sum.y) >= params.hess) {
    const float l = (left_sum.x * left_sum.x) / (left_sum.y + params.lambda);
    const float r = (right_sum.x * right_sum.x) / (right_sum.y + params.lambda);
    const float p = (total_sum.x * total_sum.x) / (total_sum.y + params.lambda);
    return l + r - p;
  } else {
    return 0.0;
  }
}

__forceinline__ __device__ __host__ float
gain_func(const float left_sum, const float total_sum, const size_t left_count,
          const size_t total_count, const GainFunctionParameters &params) {
  const size_t right_count = total_count - left_count;
  if (left_count >= params.min_leaf_size &&
      right_count >= params.min_leaf_size) {
    const float right_sum = total_sum - left_sum;
    const float l = left_sum * left_sum / (left_count + params.lambda);
    const float r = right_sum * right_sum / (right_count + params.lambda);
    const float p = total_sum * total_sum / (total_count + params.lambda);
    return l + r - p;
  } else {
    return 0.0;
  }
}

__forceinline__ __device__ __host__ float
gain_func(const double left_sum, const double total_sum,
          const size_t left_count, const size_t total_count,
          const GainFunctionParameters &params) {
  const size_t right_count = total_count - left_count;
  if (left_count >= params.min_leaf_size &&
      right_count >= params.min_leaf_size) {
    const double right_sum = total_sum - left_sum;
    const double l = left_sum * left_sum / (left_count + params.lambda);
    const double r = right_sum * right_sum / (right_count + params.lambda);
    const double p = total_sum * total_sum / (total_count + params.lambda);
    return l + r - p;
  } else {
    return 0.0;
  }
}

template <class node_type, typename sum_type>
__global__ void
gain_kernel(const sum_type *const __restrict__ left_sum,
            const float *const __restrict__ fvalues,
            const node_type *const __restrict__ segments,
            const sum_type *const __restrict__ parent_sum_iter,
            const unsigned int *const __restrict__ parent_count_iter,
            const size_t n, const GainFunctionParameters parameters,
            my_atomics *res) {
  for (unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    const float fvalue = fvalues[i + 1];
    const float fvalue_prev = fvalues[i];
    if (fvalue != fvalue_prev) {
      const node_type segment = segments[i];

      const sum_type left_sum_offset = parent_sum_iter[segment];
      const sum_type left_sum_value = left_sum[i] - left_sum_offset;

      const size_t left_count_offset = parent_count_iter[segment];
      const size_t left_count_value = i - left_count_offset;

      const sum_type total_sum = parent_sum_iter[segment + 1] - left_sum_offset;
      const size_t total_count =
          parent_count_iter[segment + 1] - left_count_offset;

      const float gain = gain_func(left_sum_value, total_sum, left_count_value,
                                   total_count, parameters);
      if (gain > 0.0) {
        updateAtomicMax(&(res[segment].ulong), gain, i);
      }
    }
  }
}

template <typename node_type, typename grad_type, typename sum_type>
class TaylorApproximationBuilder : public GardenBuilderBase {
public:
  TaylorApproximationBuilder(const TreeParam &param, const io::DataMatrix *data,
                             const InternalConfiguration &config,
                             const ApproximatedObjective<grad_type> *objective,
                             const bool verbose)
      : verbose(verbose), rnd(config.seed), overlap_depth(config.overlap),
        param(param), gain_param(param.min_leaf_size, param.min_child_weight,
                                 param.gamma, param.lambda, param.alpha),
        objective(objective),
        sparse_stat(data->columns_sparse,
                    std::vector<unsigned int>(1 << (param.depth - 2), 0)) {

    grad_d.resize(data->rows);

    active_fids.resize(data->columns);

    const int lenght = 1 << param.depth;

    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSizeGain,
                                       gain_kernel<node_type, sum_type>, 0, 0);
    gridSizeGain = (data->rows + blockSizeGain - 1) / blockSizeGain;

    minGridSize = 0;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSizeGather,
                                       gather_kernel_simple<node_type>, 0, 0);
    gridSizeGather = (data->rows + blockSizeGather - 1) / blockSizeGather;

    row2Node.resize(data->rows);
    _rowIndex2Node.resize(data->rows, 0);
    _bestSplit.resize(1 << (param.depth - 2));
    _nodeStat.resize(1 << (param.depth - 2),
                     NodeStat<sum_type>(data->columns_sparse));

    parent_node_sum.resize(lenght + 1);
    parent_node_count.resize(lenght + 1);
    parent_node_sum_h.resize(lenght + 1);
    parent_node_count_h.resize(lenght + 1);

    for (size_t i = 0; i < overlap_depth; ++i) {
      hipStream_t s;
      hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
      streams[i] = s;
      sum[i] = device_vector<sum_type>(data->rows);
      segments[i] = device_vector<node_type>(data->rows);
      segments_sorted[i] = device_vector<node_type>(data->rows);
      fvalue[i] = device_vector<float>(data->rows + 1);
      fvalue[i][0] = -std::numeric_limits<float>::infinity();
      position[i] = device_vector<unsigned int>(data->rows);
      position_sorted[i] = device_vector<unsigned int>(data->rows);
      grad_sorted[i] = device_vector<grad_type>(data->rows);
      temp_bytes_allocated[i] = 0;
      CubDebugExit(hipMalloc(&(results[i]), sizeof(my_atomics) * lenght));
      CubDebugExit(
          hipHostMalloc(&(results_h[i]), sizeof(my_atomics) * lenght));
    }
    {
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(gather_kernel_simple<float>),
                             hipFuncCachePreferL1);
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(gather_kernel_simple<node_type>),
                             hipFuncCachePreferL1);
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(gather_kernel_simple<grad_type>),
                             hipFuncCachePreferL1);
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(gather_kernel_temp<float>), hipFuncCachePreferL1);
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(gain_kernel<node_type), sum_type>,
                             hipFuncCachePreferL1);
    }
    {
      size_t max = 0;

      size_t temp_storage_bytes = 0;

      CubDebugExit(hipcub::DeviceRadixSort::SortPairs(
          NULL, temp_storage_bytes,
          thrust::raw_pointer_cast(segments[0].data()),
          thrust::raw_pointer_cast(segments_sorted[0].data()),
          thrust::raw_pointer_cast(position[0].data()),
          thrust::raw_pointer_cast(position_sorted[0].data()), data->rows, 0,
          1));

      max = std::max(max, temp_storage_bytes);

      temp_storage_bytes = 0;

      sum_type initial_value;
      init(initial_value);
      hipcub::Sum sum_op;

      CubDebugExit(hipcub::DeviceScan::ExclusiveScan(
          NULL, temp_storage_bytes,
          thrust::raw_pointer_cast(grad_sorted[0].data()),
          thrust::raw_pointer_cast(sum[0].data()), sum_op, initial_value,
          data->rows));

      max = std::max(max, temp_storage_bytes);

      temp_storage_bytes = sizeof(float) * data->rows;
      max = std::max(max, temp_storage_bytes);

      temp_storage_bytes = sizeof(grad_type) * data->rows;
      max = std::max(max, temp_storage_bytes);

      temp_storage_bytes = sizeof(node_type) * data->rows;
      max = std::max(max, temp_storage_bytes);

      for (size_t i = 0; i < overlap_depth; ++i) {
        AllocateMemoryIfRequire(i, max);
      }
    }
  }

  virtual ~TaylorApproximationBuilder() {
    for (auto i = 0; i < overlap_depth; ++i) {
      CubDebugExit(hipFree(temp_bytes[i]));
      CubDebugExit(hipFree(results[i]));
      CubDebugExit(hipHostFree(results_h[i]));
      hipStreamDestroy(streams[i]);
    }
    delete[] sum;
    delete[] segments;
    delete[] segments_sorted;
    delete[] fvalue;
    delete[] position;
    delete[] position_sorted;
    delete[] grad_sorted;
    delete[] temp_bytes;
  }

  virtual size_t MemoryRequirementsPerRecord() override {
    return (sizeof(node_type) +    // node
            sizeof(grad_type) +    // grad_sorted
            sizeof(sum_type) +     // sum
            sizeof(node_type) +    // segments
            sizeof(node_type) +    // segments_sorted
            sizeof(float) +        // fvalue
            sizeof(unsigned int) + // position
            sizeof(unsigned int)   // position sorted
            ) *
           overlap_depth;
  }

  virtual void InitGrowingTree(const size_t columns,
                               const size_t sparse_columns) override {
    int take = (int)(param.colsample_bytree * columns);
    if (take == 0) {
      printf("colsample_bytree is too small %f for %ld columns \n",
             param.colsample_bytree, columns);
      throw "colsample_bytree is too small";
    }
    take = (int)(param.colsample_bytree * param.colsample_bylevel * columns);
    if (take == 0) {
      printf("colsample_bytree and colsample_bylevel are too small %f %f for "
             "%ld columns \n",
             param.colsample_bytree, param.colsample_bylevel, columns);
      throw "colsample_bytree and colsample_bylevel are too small";
    }

    for (size_t i = 0; i < columns; ++i) {
      active_fids[i] = i;
    }

    for (size_t i = 0; i < sparse_columns; ++i) {
      std::fill(sparse_stat[i].begin(), sparse_stat[i].end(), 0);
    }

    shuffle(active_fids.begin(), active_fids.end(), rnd);

    std::fill(_rowIndex2Node.begin(), _rowIndex2Node.end(), 0);
    for (size_t i = 0; i < _nodeStat.size(); ++i) {
      _nodeStat[i].Clean();
    }
    for (size_t i = 0; i < _bestSplit.size(); ++i) {
      _bestSplit[i].Clean();
    }
  }

  virtual void InitTreeLevel(const int level, const size_t columns) override {
    int take = (int)(param.colsample_bytree * columns);
    shuffle(active_fids.begin(), active_fids.begin() + take, rnd);
  }

  virtual void GrowTree(RegTree *tree, const io::DataMatrix *data,
                        const unsigned short label) override {

    hipMemcpyAsync(
        thrust::raw_pointer_cast(grad_d.data()),
        thrust::raw_pointer_cast(objective->grad.data() + label * data->rows),
        data->rows * sizeof(grad_type), hipMemcpyHostToDevice, streams[0]);

    grad_slice = const_cast<grad_type *>(
        thrust::raw_pointer_cast(objective->grad.data() + label * data->rows));

    InitGrowingTree(data->columns, data->columns_sparse);

    for (unsigned int i = 0; (i + 1) < param.depth; ++i) {
      InitTreeLevel(i, data->columns);
      UpdateNodeStat(i, data, tree);
      FindBestSplits(i, data);
      UpdateTree(i, tree);
      UpdateNodeIndex(i, data, tree);
    }

    UpdateLeafWeight(tree);
  }

  virtual void PredictByGrownTree(RegTree *tree, io::DataMatrix *data,
                                  std::vector<float> &out) const override {
    tree->Predict(data, _rowIndex2Node, out);
  }

private:
  bool verbose;
  std::default_random_engine rnd;
  std::vector<unsigned int> active_fids;
  const unsigned short overlap_depth;
  const TreeParam param;
  const GainFunctionParameters gain_param;
  grad_type *grad_slice;
  const ApproximatedObjective<grad_type> *objective;
  host_vector<node_type,
              thrust::cuda::experimental::pinned_allocator<node_type>>
      _rowIndex2Node;
  std::vector<NodeStat<sum_type>> _nodeStat;
  std::vector<Split<sum_type>> _bestSplit;

  device_vector<sum_type> *sum = new device_vector<sum_type>[ overlap_depth ];
  device_vector<node_type> *segments =
      new device_vector<node_type>[ overlap_depth ];
  device_vector<node_type> *segments_sorted =
      new device_vector<node_type>[ overlap_depth ];
  device_vector<float> *fvalue = new device_vector<float>[ overlap_depth ];
  device_vector<unsigned int> *position =
      new device_vector<unsigned int>[ overlap_depth ];
  device_vector<unsigned int> *position_sorted =
      new device_vector<unsigned int>[ overlap_depth ];
  device_vector<grad_type> *grad_sorted =
      new device_vector<grad_type>[ overlap_depth ];
  hipStream_t *streams = new hipStream_t[overlap_depth];
  device_vector<grad_type> grad_d;
  device_vector<node_type> row2Node;
  device_vector<sum_type> parent_node_sum;
  device_vector<unsigned int> parent_node_count;
  host_vector<sum_type> parent_node_sum_h;
  host_vector<unsigned int> parent_node_count_h;
  size_t *temp_bytes_allocated = new size_t[overlap_depth];
  void **temp_bytes = new void *[overlap_depth];
  my_atomics **results = new my_atomics *[overlap_depth];
  my_atomics **results_h = new my_atomics *[overlap_depth];
  std::vector<std::vector<unsigned int>> sparse_stat;

  int blockSizeGain;
  int gridSizeGain;

  int blockSizeGather;
  int gridSizeGather;

  inline void AllocateMemoryIfRequire(const size_t circular_fid,
                                      const size_t bytes) {
    if (temp_bytes_allocated[circular_fid] == 0) {
      CubDebugExit(hipMalloc(&(temp_bytes[circular_fid]), bytes));
      temp_bytes_allocated[circular_fid] = bytes;
    } else if (temp_bytes_allocated[circular_fid] < bytes) {
      CubDebugExit(hipFree(temp_bytes[circular_fid]));
      CubDebugExit(hipMalloc(&(temp_bytes[circular_fid]), bytes));
      temp_bytes_allocated[circular_fid] = bytes;
    }
  }

  void FindBestSplits(const int level, const io::DataMatrix *data) {

    hipMemcpyAsync(thrust::raw_pointer_cast((row2Node.data())),
                    thrust::raw_pointer_cast(_rowIndex2Node.data()),
                    data->rows * sizeof(node_type), hipMemcpyHostToDevice,
                    streams[0]);

    const size_t lenght = 1 << level;

    {
      init(parent_node_sum_h[0]);
      parent_node_count_h[0] = 0;

      for (size_t i = 0; i < lenght; ++i) {
        parent_node_count_h[i + 1] =
            parent_node_count_h[i] + _nodeStat[i].count;
        parent_node_sum_h[i + 1] = parent_node_sum_h[i] + _nodeStat[i].sum_grad;
      }
      parent_node_sum = parent_node_sum_h;
      parent_node_count = parent_node_count_h;
    }

    unsigned int take = (unsigned int)(param.colsample_bylevel *
                                       param.colsample_bytree * data->columns);

    hipStreamSynchronize(streams[0]);

    for (size_t j = 0; j < take; ++j) {

      for (size_t i = 0; i < overlap_depth && (j + i) < take; ++i) {

        if (j != 0 && (i + 1) < overlap_depth) {
          continue;
        }

        size_t active_fid = active_fids[j + i];
        size_t circular_fid = (j + i) % overlap_depth;

        if (active_fid < data->columns_dense)
          ProcessDenceFeature(active_fid, circular_fid, level, data);
        else
          ProcessSparseFeature(active_fid - data->columns_dense, circular_fid,
                               level, data);
      }

      size_t circular_fid = j % overlap_depth;

      hipStream_t s = streams[circular_fid];

      hipStreamSynchronize(s);

      if (active_fids[j] < data->columns_dense) {
        GetBestSplitForDenceFeature(active_fids[j], circular_fid, lenght);
      } else {
        GetBestSplitForSparseFeature(active_fids[j] - data->columns_dense,
                                     data->columns_dense, circular_fid, lenght);
      }
    }

    for (size_t i = 0; i < lenght; ++i) {
      Split<sum_type> &split = _bestSplit[i];

      if (split.fid < 0) {
        NodeStat<sum_type> &node_stat = _nodeStat[i];
        _bestSplit[i].gain = 0.0;
        _bestSplit[i].fid = 0;
        _bestSplit[i].split_value = std::numeric_limits<float>::infinity();
        _bestSplit[i].count = node_stat.count;
        _bestSplit[i].sum_grad = node_stat.sum_grad;
      }
    }
  }

  inline void GetBestSplitForDenceFeature(const int active_fid,
                                          const size_t circular_fid,
                                          const size_t lenght) {
    for (size_t i = 0; i < lenght; ++i) {
      if (_nodeStat[i].count <= 0)
        continue;
      if (results_h[circular_fid][i].floats[0] > _bestSplit[i].gain) {
        const int index_value = results_h[circular_fid][i].ints[1];
        const sum_type s = sum[circular_fid][index_value];
        if (!_isnan(s)) {
          const float fvalue_prev_val = fvalue[circular_fid][index_value];
          const float fvalue_val = fvalue[circular_fid][index_value + 1];
          const size_t count_val =
              results_h[circular_fid][i].ints[1] - parent_node_count_h[i];

          const sum_type sum_val = s - parent_node_sum_h[i];
          _bestSplit[i].fid = active_fid;
          _bestSplit[i].gain = results_h[circular_fid][i].floats[0];
          _bestSplit[i].split_value = (fvalue_prev_val + fvalue_val) * 0.5;
          _bestSplit[i].count = count_val;
          _bestSplit[i].sum_grad = sum_val;
        } else {
          if (verbose)
            printf("sum is nan(probably infinity), consider increasing the "
                   "accuracy \n");
        }
      }
    }
  }

  inline void GetBestSplitForSparseFeature(const int active_fid,
                                           const size_t columns_dense,
                                           const size_t circular_fid,
                                           const size_t lenght) {
    for (size_t i = 0; i < lenght; ++i) {
      if (sparse_stat[active_fid][i] == 0)
        continue;
      sum_type sum_true = sum[circular_fid][i];
      if (!_isnan(sum_true)) {
        float gain = gain_func(sum_true, _nodeStat[i].sum_grad,
                               sparse_stat[active_fid][i], _nodeStat[i].count,
                               gain_param);
        if (gain > _bestSplit[i].gain) {
          _bestSplit[i].fid = active_fid + columns_dense;
          _bestSplit[i].gain = gain;
          _bestSplit[i].split_by_true = true;
          _bestSplit[i].count = sparse_stat[active_fid][i];
          _bestSplit[i].sum_grad = sum_true;
        }

      } else {
        if (verbose)
          printf("sum is nan(probably infinity), consider increasing the "
                 "accuracy \n");
      }
    }
  }

  inline void ProcessDenceFeature(const size_t active_fid,
                                  const size_t circular_fid, const int level,
                                  const io::DataMatrix *data) {

    size_t lenght = 1 << level;

    hipStream_t s = streams[circular_fid];

    device_vector<float> *fvalue_tmp = NULL;

    hipMemsetAsync(results[circular_fid], 0, lenght * sizeof(my_atomics), s);

    if (data->data_device[active_fid].size() > 0) {
      fvalue_tmp =
          const_cast<device_vector<float> *>(&(data->data_device[active_fid]));
    } else {
      hipMemcpyAsync(
          thrust::raw_pointer_cast((&fvalue[circular_fid].data()[1])),
          thrust::raw_pointer_cast(data->data[active_fid].data()),
          data->rows * sizeof(float), hipMemcpyHostToDevice, s);
      hipStreamSynchronize(s);
      fvalue_tmp = const_cast<device_vector<float> *>(&(fvalue[circular_fid]));
    }

    device_vector<unsigned int> *index_tmp = NULL;

    if (data->index_device[active_fid].size() > 0) {
      index_tmp = const_cast<device_vector<unsigned int> *>(
          &(data->index_device[active_fid]));
    } else {
      hipMemcpyAsync(thrust::raw_pointer_cast(position[circular_fid].data()),
                      thrust::raw_pointer_cast(data->index[active_fid].data()),
                      data->rows * sizeof(unsigned int), hipMemcpyHostToDevice,
                      s);
      hipStreamSynchronize(s);
      index_tmp =
          const_cast<device_vector<unsigned int> *>(&(position[circular_fid]));
    }

    gather_kernel_simple<<<gridSizeGather, blockSizeGather, 0, s>>>(
        thrust::raw_pointer_cast(index_tmp->data()),
        thrust::raw_pointer_cast(row2Node.data()),
        thrust::raw_pointer_cast(segments[circular_fid].data()), data->rows);

    size_t temp_storage_bytes = 0;

    CubDebugExit(hipcub::DeviceRadixSort::SortPairs(
        NULL, temp_storage_bytes,
        thrust::raw_pointer_cast(segments[circular_fid].data()),
        thrust::raw_pointer_cast(segments_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(index_tmp->data()),
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        data->rows, 0, level + 1, s));

    CubDebugExit(hipcub::DeviceRadixSort::SortPairs(
        temp_bytes[circular_fid], temp_storage_bytes,
        thrust::raw_pointer_cast(segments[circular_fid].data()),
        thrust::raw_pointer_cast(segments_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(index_tmp->data()),
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        data->rows, 0, level + 1, s));

    gather<<<gridSizeGather, blockSizeGather, 0, s>>>(
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(grad_d.data()),
        thrust::raw_pointer_cast(grad_sorted[circular_fid].data()), data->rows);

    gather_kernel_temp<<<gridSizeGather, blockSizeGather, 0, s>>>(
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(fvalue_tmp->data() + 1),
        thrust::raw_pointer_cast(fvalue[circular_fid].data() + 1),
        (float *)temp_bytes[circular_fid], data->rows);

    sum_type initial_value;
    init(initial_value);
    hipcub::Sum sum_op;

    temp_storage_bytes = 0;

    CubDebugExit(hipcub::DeviceScan::ExclusiveScan(
        NULL, temp_storage_bytes,
        thrust::raw_pointer_cast(grad_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(sum[circular_fid].data()), sum_op,
        initial_value, data->rows, s));

    CubDebugExit(hipcub::DeviceScan::ExclusiveScan(
        temp_bytes[circular_fid], temp_storage_bytes,
        thrust::raw_pointer_cast(grad_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(sum[circular_fid].data()), sum_op,
        initial_value, data->rows, s));

    temp_storage_bytes = 0;

    gain_kernel<<<gridSizeGain, blockSizeGain, 0, s>>>(
        thrust::raw_pointer_cast(sum[circular_fid].data()),
        thrust::raw_pointer_cast(fvalue[circular_fid].data()),
        thrust::raw_pointer_cast(segments_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(parent_node_sum.data()),
        thrust::raw_pointer_cast(parent_node_count.data()), data->rows,
        gain_param, results[circular_fid]);

    hipMemcpyAsync(results_h[circular_fid], results[circular_fid],
                    lenght * sizeof(my_atomics), hipMemcpyDeviceToHost, s);
  }

  inline void ProcessSparseFeature(const size_t active_fid,
                                   const size_t circular_fid, const int level,
                                   const io::DataMatrix *data) {
    const size_t lenght = 1 << level;
    const size_t feature_size = data->lil_column[active_fid].size();

    hipStream_t s = streams[circular_fid];

    device_vector<unsigned int> *index_tmp = NULL;

    if (data->lil_column_device[active_fid].size() > 0) {
      index_tmp = const_cast<device_vector<unsigned int> *>(
          &(data->lil_column_device[active_fid]));
    } else {
      hipMemcpyAsync(
          thrust::raw_pointer_cast(position[circular_fid].data()),
          thrust::raw_pointer_cast(data->lil_column[active_fid].data()),
          feature_size * sizeof(unsigned int), hipMemcpyHostToDevice, s);
      hipStreamSynchronize(s);
      index_tmp =
          const_cast<device_vector<unsigned int> *>(&(position[circular_fid]));
    }

    gather_kernel_simple<<<gridSizeGather, blockSizeGather, 0, s>>>(
        thrust::raw_pointer_cast(index_tmp->data()),
        thrust::raw_pointer_cast(row2Node.data()),
        thrust::raw_pointer_cast(segments[circular_fid].data()), feature_size);

    size_t temp_storage_bytes = 0;

    CubDebugExit(hipcub::DeviceRadixSort::SortPairs(
        NULL, temp_storage_bytes,
        thrust::raw_pointer_cast(segments[circular_fid].data()),
        thrust::raw_pointer_cast(segments_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(index_tmp->data()),
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        feature_size, 0, level + 1, s));

    CubDebugExit(hipcub::DeviceRadixSort::SortPairs(
        temp_bytes[circular_fid], temp_storage_bytes,
        thrust::raw_pointer_cast(segments[circular_fid].data()),
        thrust::raw_pointer_cast(segments_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(index_tmp->data()),
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        feature_size, 0, level + 1, s));

    gather<<<gridSizeGather, blockSizeGather, 0, s>>>(
        thrust::raw_pointer_cast(position_sorted[circular_fid].data()),
        thrust::raw_pointer_cast(grad_d.data()),
        thrust::raw_pointer_cast(grad_sorted[circular_fid].data()),
        feature_size);

    size_t offset = 0;
    temp_storage_bytes = 0;

    for (size_t i = 0; i < lenght; ++i) {
      if (sparse_stat[active_fid][i] == 0)
        continue;

      temp_storage_bytes = 0;

      CubDebugExit(hipcub::DeviceReduce::Sum(
          NULL, temp_storage_bytes,
          thrust::raw_pointer_cast(grad_sorted[circular_fid].data() + offset),
          thrust::raw_pointer_cast(sum[circular_fid].data() + i),
          sparse_stat[active_fid][i], s));

      CubDebugExit(hipcub::DeviceReduce::Sum(
          temp_bytes[circular_fid], temp_storage_bytes,
          thrust::raw_pointer_cast(grad_sorted[circular_fid].data() + offset),
          thrust::raw_pointer_cast(sum[circular_fid].data() + i),
          sparse_stat[active_fid][i], s));

      offset += sparse_stat[active_fid][i];
    }
  }

  void UpdateNodeStat(const int level, const io::DataMatrix *data,
                      const RegTree *tree) {
    if (level != 0) {

      const unsigned int offset = Node::HeapOffset(level);
      const unsigned int offset_next = Node::HeapOffset(level + 1);
      std::vector<NodeStat<sum_type>> tmp(
          _nodeStat.size(), NodeStat<sum_type>(data->columns_sparse));
      std::copy(_nodeStat.begin(), _nodeStat.end(), tmp.begin());

      size_t len = 1 << (level - 1);

      for (size_t i = 0; i < len; ++i) {

        _nodeStat[tree->ChildNode(i + offset, true) - offset_next].count =
            _bestSplit[i].count;
        _nodeStat[tree->ChildNode(i + offset, true) - offset_next].sum_grad =
            _bestSplit[i].sum_grad;

        _nodeStat[tree->ChildNode(i + offset, false) - offset_next].count =
            tmp[i].count - _bestSplit[i].count;

        _nodeStat[tree->ChildNode(i + offset, false) - offset_next].sum_grad =
            tmp[i].sum_grad - _bestSplit[i].sum_grad;
      }

      for (size_t i = 0; i < data->columns_sparse; ++i) {
        std::fill(sparse_stat[i].begin(), sparse_stat[i].end(), 0);
      }

      if (data->columns_sparse > 0) {
#pragma omp parallel
        {
          std::vector<std::vector<unsigned int>> temp_stat(
              data->columns_sparse,
              std::vector<unsigned int>(1 << (param.depth - 2), 0));

#pragma omp for simd
          for (size_t i = 0; i < data->rows; ++i) {
            node_type node = _rowIndex2Node[i];
            const size_t len = data->lil_row[i].size();

            for (size_t j = 0; j < len; ++j) {
              temp_stat[data->lil_row[i][j] - data->columns_dense][node] += 1;
            }
          }

#pragma omp critical
          {
            for (size_t i = 0; i < temp_stat.size(); ++i) {
              for (size_t j = 0; j < temp_stat[i].size(); ++j) {
                sparse_stat[i][j] += temp_stat[i][j];
              }
            }
          }
        }
      }

    } else {
      _nodeStat[0].count = data->rows;

      for (size_t i = 0; i < data->columns_sparse; ++i) {
        sparse_stat[i][0] = data->lil_column[i].size();
      }
      sum_type sum;
      init(sum);

#pragma omp parallel
      {
        sum_type sum_thread;
        init(sum_thread);
#pragma omp for simd
        for (size_t i = 0; i < data->rows; ++i) {
          sum_thread += grad_slice[i];
        }
#pragma omp critical
        { sum += sum_thread; }
      }
      _nodeStat[0].sum_grad = sum;
    }

    size_t len = 1 << level;

    for (size_t i = 0; i < len; ++i) {
      _nodeStat[i].gain =
          0.0; // todo: gain_func(_nodeStat[i].count, _nodeStat[i].sum_grad);
      _bestSplit[i].Clean();
    }
  }

  void UpdateTree(const int level, RegTree *tree) const {
    unsigned int offset = Node::HeapOffset(level);

    const size_t len = 1 << level;

    for (size_t i = 0; i < len; ++i) {
      const Split<sum_type> &best = _bestSplit[i];
      tree->nodes[i + offset].threshold = best.split_value;
      tree->nodes[i + offset].split_by_true = best.split_by_true;
      tree->nodes[i + offset].fid = best.fid < 0 ? 0 : best.fid;
    }
  }

  void UpdateNodeIndex(const unsigned int level, const io::DataMatrix *data,
                       RegTree *tree) {
    unsigned int const offset = Node::HeapOffset(level);
    unsigned int const offset_next = Node::HeapOffset(level + 1);

#pragma omp parallel for simd
    for (size_t i = 0; i < data->rows; ++i) {
      const unsigned int node = _rowIndex2Node[i];
      const auto &best = _bestSplit[node];
      const bool isLeft =
          (best.fid < (int)data->columns_dense &&
           data->data[best.fid][i] <= best.split_value) ||
          (best.split_by_true &&
           std::binary_search(data->lil_row[i].begin(), data->lil_row[i].end(),
                              best.fid));
      _rowIndex2Node[i] = tree->ChildNode(node + offset, isLeft) - offset_next;
    }
  }

  void UpdateLeafWeight(RegTree *tree) const {
    const unsigned int offset_1 = Node::HeapOffset(tree->depth - 2);
    const unsigned int offset = Node::HeapOffset(tree->depth - 1);
    for (unsigned int i = 0, len = (1 << (tree->depth - 2)); i < len; ++i) {
      const Split<sum_type> &best = _bestSplit[i];
      const NodeStat<sum_type> &stat = _nodeStat[i];
      tree->leaf_level[tree->ChildNode(i + offset_1, true) - offset] =
          best.LeafWeight(param) * param.eta;
      tree->leaf_level[tree->ChildNode(i + offset_1, false) - offset] =
          best.LeafWeight(stat, param) * param.eta;
    }
  }
};

Garden::Garden(const TreeParam &param, const Verbose &verbose,
               const InternalConfiguration &cfg)
    : param(param), verbose(verbose), cfg(cfg), _init(false) {}

void Garden::GrowTree(io::DataMatrix *data, float *grad) {

  if (!_init) {
    switch (param.objective) {
    case LinearRegression: {
      auto obj = new RegressionObjective(data, param.initial_y);

      if (param.depth + 1 <= sizeof(unsigned char) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned char, float, double>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned char, float, float>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned short) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned short, float, double>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned short, float, float>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned int) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned int, float, double>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder = new TaylorApproximationBuilder<unsigned int, float, float>(
              param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned long int) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned long int, float, double>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder = new TaylorApproximationBuilder<unsigned int, float, float>(
              param, data, cfg, obj, verbose.booster);
        }
      } else
        throw "unsupported depth";
      _objective = obj;
    }

    break;
    case LogisticRegression: {
      auto obj = new LogisticRegressionObjective(data, param.initial_y);

      if (param.depth + 1 <= sizeof(unsigned char) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned char, float2, mydouble2>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned char, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned short) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned short, float2, mydouble2>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned short, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned int) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned int, float2, mydouble2>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned int, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned long int) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder = new TaylorApproximationBuilder<unsigned long int, float2,
                                                    mydouble2>(
              param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned long int, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else
        throw "unsupported depth";
      _objective = obj;
    } break;
    case SoftMaxOneVsAll: {
      auto obj =
          new SoftMaxObjective(data, param.labels_count, param.initial_y);

      if (param.depth + 1 <= sizeof(unsigned char) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned char, float2, mydouble2>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned char, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned short) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned short, float2, mydouble2>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned short, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned int) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder =
              new TaylorApproximationBuilder<unsigned int, float2, mydouble2>(
                  param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned int, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      } else if (param.depth + 1 <= sizeof(unsigned long int) * CHAR_BIT) {
        if (cfg.double_precision) {
          _builder = new TaylorApproximationBuilder<unsigned long int, float2,
                                                    mydouble2>(
              param, data, cfg, obj, verbose.booster);
        } else {
          _builder =
              new TaylorApproximationBuilder<unsigned long int, float2, float2>(
                  param, data, cfg, obj, verbose.booster);
        }
      }

      else
        throw "unsupported depth";
      _objective = obj;
    } break;
    default:
      throw "Unknown objective function " + param.objective;
    }

    data->Init();

    auto mem_per_rec = _builder->MemoryRequirementsPerRecord();
    size_t total;
    size_t free;

    hipMemGetInfo(&free, &total);

    if (verbose.gpu) {
      printf("Total bytes %ld avaliable %ld \n", total, free);
      printf("Memory usage estimation %ld per record %ld in total \n",
             mem_per_rec, mem_per_rec * data->rows);
    }

    data->TransferToGPU(free * 9 / 10, verbose.gpu);

    _init = true;
  }

  if (grad == NULL) {
    _objective->UpdateGrad();
  } else {
    //          todo: fix
    //          data->grad = std::vector<float>(grad, grad + data->rows);
  }

  for (unsigned short i = 0; i < param.labels_count; ++i) {
    RegTree *tree = new RegTree(param.depth, i);
    _builder->GrowTree(tree, data, i);
    _trees.push_back(tree);
    if (grad == NULL) {
      _builder->PredictByGrownTree(tree, data, data->y_internal);
    }
  }
}

void Garden::UpdateByLastTree(io::DataMatrix *data) {
  if (data->y_internal.size() == 0)
    data->y_internal.resize(data->rows * param.labels_count,
                            _objective->IntoInternal(param.initial_y));
  for (auto it = _trees.end() - param.labels_count; it != _trees.end(); ++it) {
    (*it)->Predict(data, data->y_internal);
  }
}

void Garden::GetY(arboretum::io::DataMatrix *data,
                  std::vector<float> &out) const {
  out.resize(data->y_internal.size());
  _objective->FromInternal(data->y_internal, out);
}

void Garden::Predict(const arboretum::io::DataMatrix *data,
                     std::vector<float> &out) const {
  out.resize(data->rows * param.labels_count);
  std::vector<float> tmp(data->rows * param.labels_count);

  std::fill(tmp.begin(), tmp.end(), _objective->IntoInternal(param.initial_y));
  for (size_t i = 0; i < _trees.size(); ++i) {
    _trees[i]->Predict(data, tmp);
  }

  _objective->FromInternal(tmp, out);
}
}
}
